
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5

__global__ void vector_add(float *out, float *a, float *b, int n) {
  for (int i = 0; i < n; i++) {
    out[i] = a[i] + b[i];
  }
}

int main() {
  float *a, *b, *out;
  float *d_a;

  // Allocate memory
  a = (float *)malloc(sizeof(float) * N);
  b = (float *)malloc(sizeof(float) * N);
  out = (float *)malloc(sizeof(float) * N);

  // Allocate device memory for a
  hipMalloc((void **)&d_a, sizeof(float) * N);

  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

  // Initialize array
  for (int i = 0; i < N; i++) {
    a[i] = 1.0f;
    b[i] = 2.0f;
  }

  // Main function
  //   vector_add(out, a, b, N);
  vector_add<<<1, 1>>>(out, d_a, b, N);

  hipFree(d_a);

  // output
  for (int i = 0; i < N; i++) {
    printf("%.6f\n", *out);
  }
}
