#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _SIMPLETEXTURE3D_KERNEL_CU_
#define _SIMPLETEXTURE3D_KERNEL_CU_


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

texture<float, hipTextureType3D, hipReadModeElementType> tex;  // 3D texture
hipArray *d_volumeArray = 0;

__global__ void
d_render(float *d_output, uint imageW, uint imageH, uint imageL)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint z = blockIdx.z * blockDim.z + threadIdx.z;

    float u = (float)(x + 0.5f) / imageW;
    float v = (float)(y + 0.5f) / imageH;
    float w = (float)(z + 0.5f) / imageL;

    // read from 3D texture
    float voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH) && (z < imageL))
    {
        uint i = y * imageW + x + z * imageW * imageH;
        d_output[i] = voxel;
    }
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void freeTexture()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
    d_volumeArray = 0;
}

extern "C"
void initCuda(const float *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void *)h_volume, volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, float *d_output, uint imageW, uint imageH, uint imageL)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, imageL);
}

#endif // #ifndef _SIMPLETEXTURE3D_KERNEL_CU_
