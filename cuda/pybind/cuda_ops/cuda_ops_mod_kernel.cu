#include "hip/hip_runtime.h"
#ifndef _SIMPLETEXTURE3D_KERNEL_CU_
#define _SIMPLETEXTURE3D_KERNEL_CU_


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "cuda_ops_mod.h"

typedef unsigned int uint;
typedef unsigned char uchar;

texture<float, hipTextureType3D, hipReadModeElementType> tex;  // 3D texture
hipArray *d_volumeArray = 0;

__global__ void
d_render(float rad, float *d_output, uint imageW, uint imageH, uint imageL) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint z = blockIdx.z * blockDim.z + threadIdx.z;

    float w = (((float) z) + 0.5f) / (float) imageL;
    float u0 = (float) (x + 0.5f) - (float) imageW / 2;
    float v0 = (float) (y + 0.5f) - (float) imageH / 2;
    float u1 = u0 / (float) imageW;
    float v1 = v0 / (float) imageH;
    float tu = u1 * cosf(rad) - v1 * sinf(rad);
    float tv = v1 * cosf(rad) + u1 * sinf(rad);
    float u2 = tu + 0.5f;
    float v2 = tv + 0.5f;

    // read from 3D texture
    float voxel = tex3D(tex, u2, v2, w);

    if ((x < imageW) && (y < imageH) && (z < imageL)) {
        uint i = x + y * imageW + z * imageW * imageH;
        d_output[i] = voxel;
    }
}


__global__ void
d_render_shift(SHIFT_PARAMS shift, float *d_output, uint imageW, uint imageH, uint imageL) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint z = blockIdx.z * blockDim.z + threadIdx.z;

    float u = ((float) x + shift.x + 0.5f) / (float) imageW;
    float v = ((float) y + shift.y + 0.5f) / (float) imageH;
    float w = ((float) z + shift.z + 0.5f) / (float) imageL;

    // read from 3D texture
    float voxel;
    voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH) && (z < imageL)) {
        uint i = x + y * imageW + z * imageW * imageH;
        d_output[i] = voxel;
    }
}

__global__ void
d_render_tf2dL_z(TF_PARAMS tf_params, float *d_output, uint imageW, uint imageH, uint imageL) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint z = blockIdx.z * blockDim.z + threadIdx.z;

    // move origin to center, integer coordinates
    float w = (((float) z + 0.5f)) / (float) imageL;
    float u0 = (float) (x + 0.5f) - (float) imageW / 2;
    float v0 = (float) (y + 0.5f) - (float) imageH / 2;
    // spatial transform
    float tu = u0 * tf_params.h0 + v0 * tf_params.h1 + tf_params.h2;
    float tv = u0 * tf_params.h3 + v0 * tf_params.h4 + tf_params.h5;
    // normalize coordinates
    float u1 = tu / (float) imageW;
    float v1 = tv / (float) imageH;
    // move origin to top-left
    float u = u1 + 0.5f;
    float v = v1 + 0.5f;

    // read from 3D texture
    float voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH) && (z < imageL)) {
        uint i = x + y * imageW + z * imageW * imageH;
        d_output[i] = voxel;
    }
}

__global__ void
d_render_tf2dL_x(TF_PARAMS tf_params, float *d_output, uint imageW, uint imageH, uint imageL) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint z = blockIdx.z * blockDim.z + threadIdx.z;

    // move origin to center, integer coordinates
    float u = (float) (x + 0.5f) / (float) imageW;
    float v0 = (float) (y + 0.5f) - (float) imageH / 2;
    float w0 = (float) (z + 0.5f) - (float) imageL / 2;
    // spatial transform
    float tv = v0 * tf_params.h0 + w0 * tf_params.h1 + tf_params.h2;
    float tw = v0 * tf_params.h3 + w0 * tf_params.h4 + tf_params.h5;
    // normalize coordinates
    float v1 = tv / (float) imageH;
    float w1 = tw / (float) imageL;
    // move origin to top-left
    float v = v1 + 0.5f;
    float w = w1 + 0.5f;

    // read from 3D texture
    float voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH) && (z < imageL)) {
        uint i = x + y * imageW + z * imageW * imageH;
        d_output[i] = voxel;
    }
}

__global__ void
d_render_tf2dL_y(TF_PARAMS tf_params, float *d_output, uint imageW, uint imageH, uint imageL) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint z = blockIdx.z * blockDim.z + threadIdx.z;

    // move origin to center, integer coordinates
    float v = (float) (y + 0.5f) / (float) imageH;
    float w0 = (float) (z + 0.5f) - (float) imageL / 2;
    float u0 = (float) (x + 0.5f) - (float) imageW / 2;
    // spatial transform
    float tw = w0 * tf_params.h0 + u0 * tf_params.h1 + tf_params.h2;
    float tu = w0 * tf_params.h3 + u0 * tf_params.h4 + tf_params.h5;
    // normalize coordinates
    float w1 = tw / (float) imageL;
    float u1 = tu / (float) imageW;
    // move origin to top-left
    float w = w1 + 0.5f;
    float u = u1 + 0.5f;

    // read from 3D texture
    float voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH) && (z < imageL)) {
        uint i = x + y * imageW + z * imageW * imageH;
        d_output[i] = voxel;
    }
}

extern "C"
void setTextureFilterMode(bool bLinearFilter) {
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
//    tex.filterMode = hipFilterModeLinear;
}

extern "C"
void freeTexture() {
    checkCudaErrors(hipFreeArray(d_volumeArray));
    d_volumeArray = 0;
}

extern "C"
void initCuda(const float *h_volume, hipExtent volumeSize) {
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void *) h_volume, volumeSize.width * sizeof(float), volumeSize.width,
                                            volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent = volumeSize;
    copyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
//    tex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates
//    tex.addressMode[1] = hipAddressModeClamp;
//    tex.addressMode[2] = hipAddressModeClamp;
//    tex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
//    tex.addressMode[1] = hipAddressModeWrap;
//    tex.addressMode[2] = hipAddressModeWrap;
    tex.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, float rad, float *d_output, uint imageW, uint imageH, uint imageL) {
    d_render << < gridSize, blockSize >> > (rad, d_output, imageW, imageH, imageL);
}

extern "C"
void render_kernel_shift(dim3 gridSize, dim3 blockSize, SHIFT_PARAMS shift, float *d_output, uint imageW, uint imageH,
                         uint imageL) {
    d_render_shift << < gridSize, blockSize >> > (shift, d_output, imageW, imageH, imageL);
}

extern "C"
void render_kernel_tf2dL(dim3 gridSize, dim3 blockSize, uint axis, TF_PARAMS tf_params, float *d_output, uint imageW,
                         uint imageH,
                         uint imageL) {
    switch (axis) {
        case 0 :
            std::cout << "axis = 0\n";
            d_render_tf2dL_x << < gridSize, blockSize >> > (tf_params, d_output, imageW, imageH, imageL);
            break;       // and exits the switch
        case 1 :
            std::cout << "axis = 1\n";
            d_render_tf2dL_y << < gridSize, blockSize >> > (tf_params, d_output, imageW, imageH, imageL);
            break;       // and exits the switch
        case 2 :
            std::cout << "axis = 2\n";
            d_render_tf2dL_z << < gridSize, blockSize >> > (tf_params, d_output, imageW, imageH, imageL);
            break;
        default: // compilation error: jump to default: would enter the scope of 'x'
            std::cerr << "Error: axis not in [0,1,2] !!!\n";
            break;
    }
}

#endif // #ifndef _SIMPLETEXTURE3D_KERNEL_CU_
