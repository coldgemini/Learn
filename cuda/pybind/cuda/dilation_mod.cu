#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>


__global__ void dilation_ker(bool *out, bool *a, int X, int Y, int Z) {
    // block parallel version
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int tid = x + y * Y + z * Z * Y;

    if (a[tid])
        out[tid] = 1;
    else
        return;

    if (x - 1 >= 0) {
        int x_t = x - 1;
        int oid = x_t + y * Y + z * Z * Y;
        out[oid] = 1;
    }
    if (x + 1 <= X) {
        int x_t = x + 1;
        int oid = x_t + y * Y + z * Z * Y;
        out[oid] = 1;
    }
    if (y - 1 >= 0) {
        int y_t = y - 1;
        int oid = x + y_t * Y + z * Z * Y;
        out[oid] = 1;
    }
    if (y + 1 <= Y) {
        int y_t = y + 1;
        int oid = x + y_t * Y + z * Z * Y;
        out[oid] = 1;
    }
    if (z - 1 >= 0) {
        int z_t = z - 1;
        int oid = x + y * Y + z_t * Z * Y;
        out[oid] = 1;
    }
    if (z + 1 <= Z) {
        int z_t = z + 1;
        int oid = x + y * Y + z_t * Z * Y;
        out[oid] = 1;
    }
}

namespace py = pybind11;


py::array_t<bool> dilation(py::array_t<bool> input1) {

/*  read input arrays buffer_info */
    auto buf1 = input1.request();
//    if (buf1.size != buf2.size)
//        throw std::runtime_error("Input shapes must match");

/*  allocate the output buffer */
    py::array_t<bool> result = py::array_t<bool>(buf1.size);
    auto buf3 = result.request();

    bool *ptr1 = (bool *) buf1.ptr, *ptr3 = (bool *) buf3.ptr;
    size_t X = buf1.shape[0];
    size_t Y = buf1.shape[1];
    size_t Z = buf1.shape[2];
    std::cout << "X: " << X << std::endl;
    std::cout << "Y: " << Y << std::endl;
    std::cout << "Z: " << Z << std::endl;


    bool *d_a, *d_out;


    // Allocate device memory
    hipMalloc((void **) &d_a, sizeof(bool) * X * Y * Z);
    hipMalloc((void **) &d_out, sizeof(bool) * X * Y * Z);

    // Transfer data from host to device memory
    hipMemcpy(d_a, ptr1, sizeof(bool) * X * Y * Z, hipMemcpyHostToDevice);

    // Executing kernel
//    vector_add2 << < 1, X * Y >> > (d_out, d_a, d_b);
//    vector_add2 << < X, Y >> > (d_out, d_a, d_b);
    dim3 grid(1, 1, 1);
    dim3 block(X, Y, Z);
//    vector_add3 << < grid, block >> > (d_out, d_a, d_b, X);
//    vector_add3 << < grid, block >> > (d_out, d_a, d_b, Y);
    dilation_ker << < grid, block >> > (d_out, d_a, X, Y, Z);

    // Transfer data back to host memory
    hipMemcpy(ptr3, d_out, sizeof(bool) * X * Y * Z, hipMemcpyDeviceToHost);


    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_out);


// reshape result
    result.resize({X, Y, Z});

    return result;
}


PYBIND11_MODULE(dilation_mod, m
) {
m.

doc() = "dilation using cuda pybind11"; // optional module docstring
m.def("dilation", &dilation, "dilation");
}
