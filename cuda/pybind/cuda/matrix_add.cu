#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N 4
#define BLOCK_DIM 4

__global__ void matrixAdd(int *a, int *b, int *c);

int main() {
    int a[N][N], b[N][N], c[N][N];
    int *dev_a, *dev_b, *dev_c;
    int size = N * N * sizeof(int);

    for (size_t idx = 0; idx < N; idx++)
        for (size_t idy = 0; idy < N; idy++) {
            a[idx][idy] = idx;
            b[idx][idy] = idy;
        }
    // initialize a and b with real values (NOT SHOWN)
    hipMalloc((void **) &dev_a, size);
    hipMalloc((void **) &dev_b, size);
    hipMalloc((void **) &dev_c, size);
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((int) ceil(N / dimBlock.x), (int) ceil(N / dimBlock.y));
    matrixAdd << < dimGrid, dimBlock >> > (dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (size_t idx = 0; idx < N; idx++)
        for (size_t idy = 0; idy < N; idy++)
            std::cout << c[idx][idy] << std::endl;
}

__global__ void matrixAdd(int *a, int *b, int *c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * N;
    if (col < N && row < N) {
        c[index] = a[index] + b[index];
    }
}