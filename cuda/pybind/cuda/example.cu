#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>


__global__ void vector_add(float *out, float *a, float *b, int n) {
    // serial version
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_add2(float *out, float *a, float *b) {
    // linear parallel version
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 4)
        out[tid] = a[tid] + b[tid];
}

__global__ void vector_add3(float *out, float *a, float *b, int n) {
    // block parallel version
//    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = x + y * n;
    if (tid < 10)
//        out[tid] = a[tid] + b[tid];
        out[tid] = tid;
}

namespace py = pybind11;


py::array_t<float> add_arrays(py::array_t<float> input1, py::array_t<float> input2) {

/*  read input arrays buffer_info */
    auto buf1 = input1.request(), buf2 = input2.request();
    if (buf1.size != buf2.size)
        throw std::runtime_error("Input shapes must match");

/*  allocate the output buffer */
    py::array_t<float> result = py::array_t<float>(buf1.size);
    auto buf3 = result.request();

    float *ptr1 = (float *) buf1.ptr, *ptr2 = (float *) buf2.ptr, *ptr3 = (float *) buf3.ptr;
    size_t X = buf1.shape[0];
    size_t Y = buf1.shape[1];
    std::cout << "X: " << X << std::endl;
    std::cout << "Y: " << Y << std::endl;


    float *d_a, *d_b, *d_out;


    // Allocate device memory
    hipMalloc((void **) &d_a, sizeof(float) * X * Y);
    hipMalloc((void **) &d_b, sizeof(float) * X * Y);
    hipMalloc((void **) &d_out, sizeof(float) * X * Y);

    // Transfer data from host to device memory
    hipMemcpy(d_a, ptr1, sizeof(float) * X * Y, hipMemcpyHostToDevice);
    hipMemcpy(d_b, ptr2, sizeof(float) * X * Y, hipMemcpyHostToDevice);

    // Executing kernel
//    vector_add2 << < 1, X * Y >> > (d_out, d_a, d_b);
//    vector_add2 << < X, Y >> > (d_out, d_a, d_b);
    dim3 grid(2, 2);
    dim3 block(X / 2, Y / 2);
    vector_add3 << < grid, block >> > (d_out, d_a, d_b, X);
//    vector_add3 << < grid, block >> > (d_out, d_a, d_b, Y);

    // Transfer data back to host memory
    hipMemcpy(ptr3, d_out, sizeof(float) * X * Y, hipMemcpyDeviceToHost);


    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);


// reshape result
    result.resize({X, Y});

    return result;
}


PYBIND11_MODULE(example, m
) {
m.

doc() = "Add two vectors using pybind11"; // optional module docstring
m.def("add_arrays", &add_arrays, "Add two NumPy arrays");
}
