#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

////////////////////////////////////////////////////////////////////////////////
// Constants
const float angle = 0.5f;        // angle to rotate image by (in radians)

// Texture reference for 2D float texture
texture<float, 1, hipReadModeElementType> tex;

// Auto-Verification Code
bool testResult = true;


__global__ void interp1d_ker(float *out, float *a, int X) {
    // block parallel version
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = x + y * Y + z * Z * Y;

}

namespace py = pybind11;


py::array_t<float> interp1d(py::array_t<float> input1) {

/*  read input arrays buffer_info */
    auto bufin = input1.request();
//    if (buf1.size != buf2.size)
//        throw std::runtime_error("Input shapes must match");

/*  allocate the output buffer */
    py::array_t<float> result = py::array_t<float>(bufin.size);
    auto bufout = result.request();
    float *ptr1 = (float *) bufin.ptr, *ptr3 = (float *) bufout.ptr;
    size_t X = bufin.shape[0];
    std::cout << "X: " << X << std::endl;

    float *d_a, *d_out;

    // Allocate device memory
    hipMalloc((void **) &d_a, sizeof(float) * X);
    hipMalloc((void **) &d_out, sizeof(float) * X);

    // Transfer data from host to device memory
    hipMemcpy(d_a, ptr1, sizeof(float) * X, hipMemcpyHostToDevice);

    // Executing kernel
//    vector_add2 << < 1, X * Y >> > (d_out, d_a, d_b);
//    vector_add2 << < X, Y >> > (d_out, d_a, d_b);
    dim3 grid(1, 1, 1);
    dim3 block(X, Y, Z);
//    vector_add3 << < grid, block >> > (d_out, d_a, d_b, X);
//    vector_add3 << < grid, block >> > (d_out, d_a, d_b, Y);
    dilation_ker << < grid, block >> > (d_out, d_a, X, Y, Z);

    // Transfer data back to host memory
    hipMemcpy(ptr3, d_out, sizeof(float) * X * Y * Z, hipMemcpyDeviceToHost);


    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_out);


// reshape result
    result.resize({X, Y, Z});

    return result;
}


PYBIND11_MODULE(dilation_mod, m
) {
m.

doc() = "dilation using cuda pybind11"; // optional module docstring
m.def("dilation", &dilation, "dilation");
}
