#include "hip/hip_runtime.h"
//
// Created by xiang on 19-7-9.
//

#include <cstdio>
//#include "handle.h"
#include <hip/hip_runtime_api.h>

using namespace std;

texture<float, 2> tex_w;

__global__ void kernel(int imax, float(*w)[3], float (*f)[3]) {
    int i = threadIdx.x;
    int j = threadIdx.y;

    if (i < imax)
        f[i][j] = tex2D(tex_w, i, j);
}

void print_to_stdio(int imax, float (*w)[3]) {
    for (int i = 0; i < imax; i++) {
        printf("%2d  %3.6f\t  %3.6f\t %3.6f\n", i, w[i][0], w[i][1], w[i][2]);
    }
}

int main(void) {
    int imax = 8;
    float (*w)[3];
    float (*d_w)[3], (*d_f)[3];
    dim3 grid(imax, 3);

    w = (float (*)[3]) malloc(imax * 3 * sizeof(float));

    for (int i = 0; i < imax; i++) {
        for (int j = 0; j < 3; j++) {
            w[i][j] = i + 0.01f * j;
        }
    }

    print_to_stdio(imax, w);

    hipMalloc((void **) &d_w, 3 * imax * sizeof(float));
    hipMalloc((void **) &d_f, 3 * imax * sizeof(float));

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
//    HANDLE_ERROR(hipBindTexture2D(NULL, tex_w, d_w, desc, imax, 3, sizeof(float) * imax));
    checkCudaErrors(hipBindTexture2D(NULL, tex_w, d_w, desc, imax, 3, sizeof(float) * imax));

    hipMemcpy(d_w, w, 3 * imax * sizeof(float), hipMemcpyHostToDevice);

    // just use threads for simplicity
    kernel << < 1, grid >> > (imax, d_w, d_f);

    hipMemcpy(w, d_f, 3 * imax * sizeof(float), hipMemcpyDeviceToHost);

    hipUnbindTexture(tex_w);
    hipFree(d_w);
    hipFree(d_f);

    print_to_stdio(imax, w);

    free(w);
    return 0;
}